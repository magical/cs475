#include "hip/hip_runtime.h"
// Array multiplication: C = A * B:
// vim: syntax=cuda noet sw=8

// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
//#include "helper_functions.h"
#include "hip/hip_runtime_api.h"


#ifndef BLOCKSIZE
#define BLOCKSIZE		32		// number of threads per block
#endif

#ifndef SIZE
#define SIZE			1*1024*1024	// array size
#endif


__global__ void AutoCorrelate(float *dArray, float *dSums, int size) {
	int gid = blockIdx.x*blockDim.x + threadIdx.x;

	int shift = gid;
	float sum = 0.;
	for (int i = 0; i < size; i++) {
		sum += dArray[i] * dArray[i + shift];
	}
	dSums[shift] = sum;
}


// main program:

int
main( int argc, char* argv[ ] )
{
	int dev = findCudaDevice(argc, (const char **)argv);

	// allocate host memory:

	float * hA = new float [ SIZE ];
	float * hB = new float [ SIZE ];
	float * hC = new float [ SIZE/BLOCKSIZE ];

	for( int i = 0; i < SIZE; i++ )
	{
		hA[i] = hB[i] = (float) sqrt(  (float)(i+1)  );
	}

	// allocate device memory:

	float *dA, *dB, *dC;

	dim3 dimsA( SIZE, 1, 1 );
	dim3 dimsB( SIZE, 1, 1 );
	dim3 dimsC( SIZE/BLOCKSIZE, 1, 1 );

	//__shared__ float prods[SIZE/BLOCKSIZE];


	checkCudaErrors(hipMalloc( reinterpret_cast<void **>(&dA), SIZE*sizeof(float) ));
	checkCudaErrors(hipMalloc( reinterpret_cast<void **>(&dB), SIZE*sizeof(float) ));
	checkCudaErrors(hipMalloc( reinterpret_cast<void **>(&dC), (SIZE/BLOCKSIZE)*sizeof(float) ));


	// copy host memory to the device:

	checkCudaErrors(hipMemcpy( dA, hA, SIZE*sizeof(float), hipMemcpyHostToDevice ));
	checkCudaErrors(hipMemcpy( dB, hB, SIZE*sizeof(float), hipMemcpyHostToDevice ));

	// setup the execution parameters:

	dim3 threads(BLOCKSIZE, 1, 1 );
	dim3 grid( SIZE / threads.x, 1, 1 );

	// Create and start timer

	hipDeviceSynchronize( );

	// allocate CUDA events that we'll use for timing:

	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate( &start ));
	checkCudaErrors(hipEventCreate( &stop ));

	// record the start event:

	checkCudaErrors(hipEventRecord( start, NULL ));

	// execute the kernel:

	AutoCorrelate<<< grid, threads >>>( dA, dB, dC );

	// record the stop event:

	checkCudaErrors(hipEventRecord( stop, NULL ));

	// wait for the stop event to complete:

	checkCudaErrors(hipEventSynchronize( stop ));

	float msecTotal = 0.0f;
	checkCudaErrors(hipEventElapsedTime( &msecTotal, start, stop ));

	// compute and print the performance

	double secondsTotal = 0.001 * (double)msecTotal;
	double multsPerSecond = (float)SIZE / secondsTotal;
	double megaMultsPerSecond = multsPerSecond / 1000000.;
	fprintf( stderr, "Array Size = %10d, MegaMultReductions/Second = %10.2lf\n", SIZE, megaMultsPerSecond );

	// copy result from the device to the host:

	checkCudaErrors(hipMemcpy( hC, dC, (SIZE/BLOCKSIZE)*sizeof(float), hipMemcpyDeviceToHost ));

	// clean up memory:
	delete [ ] hA;
	delete [ ] hB;
	delete [ ] hC;

	checkCudaErrors(hipFree( dA ));
	checkCudaErrors(hipFree( dB ));
	checkCudaErrors(hipFree( dC ));


	return 0;
}

