#include "hip/hip_runtime.h"
// Array multiplication: C = A * B:
// vim: syntax=cuda noet sw=8

// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
//#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

#include "read.h"

const int BLOCKSIZE = 256; // number of threads per block


__global__ void AutoCorrelate(float *dArray, float *dSums, int size) {
	int gid = blockIdx.x*blockDim.x + threadIdx.x;

	int shift = gid;
	float sum = 0.;
	for (int i = 0; i < size; i++) {
		sum += dArray[i] * dArray[i + shift];
	}
	dSums[shift] = sum;
}


// main program:

int
main( int argc, char* argv[ ] )
{
	int dev = findCudaDevice(argc, (const char **)argv);

	// read input data & allocate host memory:

	float *hArray = NULL;
	int size = 0;
	ReadData("signal.txt", &hArray, &size);

	float *hSums = new float[size];

	// allocate device memory:

	float *dArray, *dSums;

	checkCudaErrors(hipMalloc( reinterpret_cast<void **>(&dArray), 2*size*sizeof(float) ));
	checkCudaErrors(hipMalloc( reinterpret_cast<void **>(&dSums), size*sizeof(float) ));

	// copy host memory to the device:

	checkCudaErrors(hipMemcpy( dArray, hArray, 2*size*sizeof(float), hipMemcpyHostToDevice ));

	// setup the execution parameters:

	dim3 threads(BLOCKSIZE, 1, 1 );
	dim3 grid( size / threads.x, 1, 1 );

	// Create and start timer

	hipDeviceSynchronize( );

	// allocate CUDA events that we'll use for timing:

	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate( &start ));
	checkCudaErrors(hipEventCreate( &stop ));

	// record the start event:

	checkCudaErrors(hipEventRecord( start, NULL ));

	// execute the kernel:

	AutoCorrelate<<< grid, threads >>>( dArray, dSums, size );

	// record the stop event:

	checkCudaErrors(hipEventRecord( stop, NULL ));

	// wait for the stop event to complete:

	checkCudaErrors(hipEventSynchronize( stop ));

	float msecTotal = 0.0f;
	checkCudaErrors(hipEventElapsedTime( &msecTotal, start, stop ));

	// compute and print the performance

	double secondsTotal = 0.001 * (double)msecTotal;
	double multsPerSecond = (float)size / secondsTotal;
	double kiloMultsPerSecond = multsPerSecond / 1e3;
	fprintf( stderr, "Array Size = %10d, KiloAutoCorrelates/Second = %10.2lf\n", size, kiloMultsPerSecond );

	// copy result from the device to the host:

	checkCudaErrors(hipMemcpy( hSums, dSums, size*sizeof(float), hipMemcpyDeviceToHost ));

	for (int i = 1; i <= 512; i++ ) {
		printf("%d %f\n", i, hSums[i]);
	}

	// clean up memory:
	delete [ ] hArray;
	delete [ ] hSums;

	checkCudaErrors(hipFree( dArray ));
	checkCudaErrors(hipFree( dSums ));


	return 0;
}

