#include "hip/hip_runtime.h"
// Monte carlo beam simulation

// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

// ranges for the random numbers:
const float XCMIN =	 0.0;
const float XCMAX =	+2.0;
const float YCMIN =	 0.0;
const float YCMAX =	 2.0;
const float RMIN  =	 0.5;
const float RMAX  =	 2.0;


#ifndef BLOCKSIZE
#define BLOCKSIZE		32		// number of threads per block
#endif

#ifndef SIZE
#define SIZE			1*1024*1024	// array size
#endif

#ifndef TOLERANCE
#define TOLERANCE		0.00001f	// tolerance to relative error
#endif

float Ranf( float, float );

// array multiplication (CUDA Kernel) on the device: C = A * B

__global__  void ArrayMul( float *A, float *B, float *C )
{
	__shared__ float prods[BLOCKSIZE];

	unsigned int numItems = blockDim.x;
	unsigned int tnum = threadIdx.x;
	unsigned int wgNum = blockIdx.x;
	unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;

	prods[tnum] = A[gid] * B[gid];

	for (int offset = 1; offset < numItems; offset *= 2)
	{
		int mask = 2 * offset - 1;
		__syncthreads();
		if ((tnum & mask) == 0)
		{
			prods[tnum] += prods[tnum + offset];
		}
	}

	__syncthreads();
	if (tnum == 0)
		C[wgNum] = prods[0];
}

// monte carlo
__global__ void MonteCarlo( float *xcs, float *ycs, float *rs, int *hits )
{
	unsigned int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;

	// randomize the location and radius of the circle:
	float xc = xcs[globalThreadId];
	float yc = ycs[globalThreadId];
	float  r =  rs[globalThreadId];

	// solve for the intersection using the quadratic formula:
	float a = 2.;
	float b = -2.*( xc + yc );
	float c = xc*xc + yc*yc - r*r;
	float d = b*b - 4.*a*c;

	// If d is less than 0., then the circle was completely missed. (Case A) Doesn't hit.
	if (d < 0) {
		return;
	}

	// hits the circle:
	// get the first intersection:
	d = sqrt( d );
	float t1 = (-b + d ) / ( 2.*a );	// time to intersect the circle
	float t2 = (-b - d ) / ( 2.*a );	// time to intersect the circle
	float tmin = t1 < t2 ? t1 : t2;		// only care about the first intersection

	// If tmin is less than 0., then the circle completely engulfs the laser pointer. (Case B) Doesn't hit.
	if (tmin < 0) {
		return;
	}

	// where does it intersect the circle?
	float xcir = tmin;
	float ycir = tmin;

	// get the unitized normal vector at the point of intersection:
	float nx = xcir - xc;
	float ny = ycir - yc;
	float n = sqrt( nx*nx + ny*ny );
	nx /= n;	// unit vector
	ny /= n;	// unit vector

	// get the unitized incoming vector:
	float inx = xcir - 0.;
	float iny = ycir - 0.;
	float in = sqrt( inx*inx + iny*iny );
	inx /= in;	// unit vector
	iny /= in;	// unit vector

	// get the outgoing (bounced) vector:
	float dot = inx*nx + iny*ny;
	float outx = inx - 2.*nx*dot;	// angle of reflection = angle of incidence`
	float outy = iny - 2.*ny*dot;	// angle of reflection = angle of incidence`

	// find out if it hits the infinite plate:
	float t = ( 0. - ycir ) / outy;

	// If t is less than 0., then the reflected beam went up instead of down. Doesn't hit.
	if (t < 0) {
		return;
	}

	// Otherwise, this beam hit the infinite plate. (Case D)
	hits[globalThreadId] = 1;

}

float Ranf( float low, float high )
{
	float r = (float) rand();		// 0 - RAND_MAX
	float t = r  / (float) RAND_MAX;	// 0. - 1.

	return low  +  t * ( high - low );
}

// main program:

int
main( int argc, char* argv[ ] )
{
	int dev = findCudaDevice(argc, (const char **)argv);

	// allocate host memory:
	int numtrials = SIZE;

	// better to define these here so that the rand() calls don't get into the thread timing:
	float *xcs = new float[ numtrials ];
	float *ycs = new float[ numtrials ];
	float * rs = new float[ numtrials ];
	int * hits = new int[ numtrials ]; // TODO: int8_t?

	// fill the random-value arrays:
	for( int n = 0; n < numtrials; n++ )
	{
		xcs[n] = Ranf( XCMIN, XCMAX );
		ycs[n] = Ranf( YCMIN, YCMAX );
		rs[n] = Ranf(  RMIN,  RMAX );
		hits[n] = 0;
	}

	// allocate device memory:

	float *dXcs, *dYcs, *dRs;
	int *dHits;

	//dim3 dimsA( SIZE, 1, 1 );
	//dim3 dimsB( SIZE, 1, 1 );
	//dim3 dimsC( SIZE/BLOCKSIZE, 1, 1 );

	//__shared__ float prods[SIZE/BLOCKSIZE];


	hipError_t status;
	status = hipMalloc( reinterpret_cast<void **>(&dXcs), numtrials*sizeof(float) );
	checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dYcs), numtrials*sizeof(float) );
	checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dRs), numtrials*sizeof(float) );
	checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dHits), numtrials*sizeof(int) );
	checkCudaErrors( status );


	// copy host memory to the device:

	status = hipMemcpy( dXcs, xcs, numtrials*sizeof(float), hipMemcpyHostToDevice );
	checkCudaErrors( status );
	status = hipMemcpy( dYcs, ycs, numtrials*sizeof(float), hipMemcpyHostToDevice );
	checkCudaErrors( status );
	status = hipMemcpy( dRs, rs, numtrials*sizeof(float), hipMemcpyHostToDevice );
	checkCudaErrors( status );

	// setup the execution parameters:

	dim3 threads(BLOCKSIZE, 1, 1 );
	dim3 grid( numtrials / threads.x, 1, 1 );

	// Create and start timer

	hipDeviceSynchronize( );

	// allocate CUDA events that we'll use for timing:

	hipEvent_t start, stop;
	status = hipEventCreate( &start );
		checkCudaErrors( status );
	status = hipEventCreate( &stop );
		checkCudaErrors( status );

	// record the start event:

	status = hipEventRecord( start, NULL );
		checkCudaErrors( status );

	// execute the kernel:

	MonteCarlo<<< grid, threads >>>( dXcs, dYcs, dRs, dHits );

	// record the stop event:

	status = hipEventRecord( stop, NULL );
		checkCudaErrors( status );

	// wait for the stop event to complete:

	status = hipEventSynchronize( stop );
		checkCudaErrors( status );

	float msecTotal = 0.0f;
	status = hipEventElapsedTime( &msecTotal, start, stop );
		checkCudaErrors( status );

	// compute and print the performance

	double secondsTotal = 0.001 * (double)msecTotal;
	double multsPerSecond = (float)numtrials / secondsTotal;
	double megaMultsPerSecond = multsPerSecond / 1000000.;
	fprintf( stderr, "Array Size = %10d, MegaMultReductions/Second = %10.2lf\n", numtrials, megaMultsPerSecond );

	// copy result from the device to the host:

	status = hipMemcpy( hits, dHits, numtrials*sizeof(int), hipMemcpyDeviceToHost );
	checkCudaErrors( status );

	int numHits = 0;
	for (int i = 0; i < numtrials; i++) {
		numHits += hits[i];
	}

	printf("Num hits = %d, hit rate = %.4f\n", numHits, (double)numHits / numtrials);

	/* 
	status = hipMemcpy( hC, dC, (SIZE/BLOCKSIZE)*sizeof(float), hipMemcpyDeviceToHost );
		checkCudaErrors( status );

	// check the sum :

	double sum = 0.;
	for(int i = 0; i < SIZE/BLOCKSIZE; i++ )
	{
		//fprintf(stderr, "hC[%6d] = %10.2f\n", i, hC[i]);
		sum += (double)hC[i];
	}
	fprintf( stderr, "\nsum = %10.2lf\n", sum );
	*/

	// clean up memory:
	delete [ ] xcs;
	delete [ ] ycs;
	delete [ ] rs;
	delete [ ] hits;

	status = hipFree( dXcs ); checkCudaErrors( status );
	status = hipFree( dYcs ); checkCudaErrors( status );
	status = hipFree( dRs ); checkCudaErrors( status );
	status = hipFree( dHits ); checkCudaErrors( status );


	return 0;
}

